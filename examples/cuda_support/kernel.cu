#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


/*!	\def ConvertSMVer2Cores(major, minor)
	\brief Get number of CUDA cores per multiprocessor.
	\link https://en.wikipedia.org/wiki/CUDA
	\arg[in] major GPU Architecture major version.
	\arg[in] minor GPU Architecture minor version.
	\returns 0 if GPU Architecture is unknown, or number of CUDA cores per multiprocessor.
*/
#define ConvertSMVer2Cores(major, minor) \
	(((major) == 1)? ( /* Tesla */ \
		((minor) == 0)? 8: /* G80*/ \
		((minor) == 1)? 8: /* G8x, G9x */ \
		((minor) == 2)? 8: /* GT21x */ \
		((minor) == 3)? 8: /* GT200 */ \
		0): \
	((major) == 2)? ( /* Fermi */ \
		((minor) == 0)? 32: /* GF100, GF110 */ \
		((minor) == 1)? 48: /* GF10x, FG11x */ \
		0): \
	((major) == 3)? ( /* Kepler */ \
		((minor) == 0)? 192: /* GK10x */ \
		((minor) == 2)? 192: /* GK20A */ \
		((minor) == 5)? 192: /* GK11x, GK208 */ \
		((minor) == 7)? 192: /* GK210 */ \
		0): \
	((major) == 5)? ( /* Maxwell */ \
		((minor) == 0)? 128: /* GM10X */ \
		((minor) == 2)? 128: /* GM20X */ \
		((minor) == 3)? 128: /* GM20B */ \
		0): \
	((major) == 6)? ( /* Pascal */ \
		((minor) == 0)? 64:  /* GP100 */ \
		((minor) == 1)? 128: /* GP10X */ \
		((minor) == 2)? 128: /* GP10B */ \
		0): \
	((major) == 7)? ( /* Volta */ \
		((minor) == 0)? 64:  /* GV100 */ \
		0): \
	0)

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

	int count = 0;
	if (hipGetDeviceCount(&count) != hipSuccess) return 0;
	printf("device: %d\n", count);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("name: %s\n", prop.name);
	printf("compute capability: %d.%d\n", prop.major, prop.minor);
	printf("multiProcessorCount: %d (%d Cores)\n", prop.multiProcessorCount, 
		ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount);
	printf("maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
	printf("maxThreadsPerMultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
	printf("warpSize: %d\n", prop.warpSize);
	printf("maxThreadsDim: %d, %d, %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("totalGlobalMem: %zu MiB\n", prop.totalGlobalMem/1024/1024);
	printf("clockRate: %d MHz\n", prop.clockRate/1000);
	printf("isMultiGpuBoard: %d\n", prop.isMultiGpuBoard);
	printf("maxGridSize: %d x %d x %d\n", prop.maxGridSize[0], 
		prop.maxGridSize[1], 
		prop.maxGridSize[2]);


    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
